#include "hip/hip_runtime.h"
#include <unistd.h>
#include <iostream>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "../fp16_conversion.h"

using namespace std;

const char* cublasGetErrorString(hipblasStatus_t status)
{
    switch(status)
    {
        case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
        case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
        case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
        case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
        case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
        case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
        case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
        case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
        case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED";
	case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN";
    }
    return "unknown error";
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

inline
hipblasStatus_t checkCublas(hipblasStatus_t result)
{
  if (result != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", cublasGetErrorString(result));
    assert(result == HIPBLAS_STATUS_SUCCESS);
  }
  return result;
}

// Fill the array A(nr_rows_A, nr_cols_A) with random numbers on CPU
void CPU_fill_rand(float *A, int nr_rows_A, int nr_cols_A) {
	int a=1;

    for(int i = 0; i < nr_rows_A * nr_cols_A; i++){
          A[i] = (float)rand()/(float)(RAND_MAX/a);
	}
}

int main(int argc, char ** argv){


  int min_m_k_n = 1024;
  int max_m_k_n = 16384*2;
  int repeats = 100;
  int verbose = 1;

  if(verbose) 
    cout << "running with" 
	 << " min_m_k_n: " << min_m_k_n
	 << " max_m_k_n: " << max_m_k_n
	 << " repeats: " << repeats
	 << endl;

  hipblasStatus_t stat;
  hipblasHandle_t handle;

  checkCublas(hipblasCreate(&handle));

  if(verbose) cout << "allocating device variables" << endl;
  
  // Allocate 3 arrays on CPU
  
  float *h_A = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  float *h_B = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));
  float *h_C = (float *)malloc(max_m_k_n * max_m_k_n * sizeof(float));

  CPU_fill_rand(h_A, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_B, max_m_k_n, max_m_k_n);
  CPU_fill_rand(h_C, max_m_k_n, max_m_k_n);
  
  // Allocate 3 arrays on GPU
  hip_bfloat16 *d_A, *d_B, *d_C;
  checkCuda(hipMallocManaged(&d_A, max_m_k_n * max_m_k_n * sizeof(hip_bfloat16)));
  checkCuda(hipMallocManaged(&d_B, max_m_k_n * max_m_k_n * sizeof(hip_bfloat16)));
  checkCuda(hipMallocManaged(&d_C, max_m_k_n * max_m_k_n * sizeof(hip_bfloat16)));    
  
  for (int i = 0; i < max_m_k_n * max_m_k_n; i++) {
    d_A[i] = __float2bfloat16(h_A[i]);
    d_B[i] = __float2bfloat16(h_B[i]);
    d_C[i] = __float2bfloat16(h_C[i]);
  }

  int lda, ldb, ldc, m, n, k;
  const hip_bfloat16 alf = __float2bfloat16(1.0f);
  const hip_bfloat16 bet = __float2bfloat16(0.0f);
  const hip_bfloat16 *alpha = &alf;
  const hip_bfloat16 *beta = &bet;
  
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  for(int size = min_m_k_n; size <= max_m_k_n; size=size*2){
    double sum = 0.0;
    for(int rep = 0; rep < repeats; rep++){
      hipEventRecord(start, 0);
	  m=n=k=size;
	  lda = m;
	  ldb = k;
	  ldc = m;
      
      stat = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, d_A, HIP_R_16BF, lda, d_B, HIP_R_16BF, ldb, beta, d_C, HIP_R_16BF, ldc, HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT);

      hipEventRecord(stop,0);
      hipEventSynchronize(stop);
      if(stat != HIPBLAS_STATUS_SUCCESS){
	  fprintf(stderr, "CuBLAS Error: %s\n", cublasGetErrorString(stat));
          exit(1);
      }
      assert(!hipGetLastError());
      
      float elapsed;
      hipEventElapsedTime(&elapsed, start, stop);
      elapsed /= 1000.0f;
      if (rep >= 25) {
          sum += elapsed;
      }
    }
  cout << "bfloat16: size " 
  << size << " average: " << sum/75 << " s "<< endl;

  }

  //Free GPU memory
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  // Free CPU memory
  free(h_A);
  free(h_B);
  free(h_C);
      
  return 0;
}
